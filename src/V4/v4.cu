#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hipblas.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 128
#define OUTPUT_SIZE 10
#define LEARNING_RATE 0.007f
#define EPOCHS 3
#define BATCH_SIZE 32
#define NUM_CLASSES 10
#define BLOCK_SIZE 32

__constant__ float input[INPUT_SIZE];

float get_time(clock_t start) {
    return (float)(clock() - start) / CLOCKS_PER_SEC;
}

__device__ float sigmoid(float x) {
    return 1.0f / (1.0f + expf(-x));
}

float** allocateMatrix(int rows, int cols) {
    float** mat = (float**)malloc(rows * sizeof(float*));
    for (int i = 0; i < rows; i++) {
        mat[i] = (float*)malloc(cols * sizeof(float));
    }
    return mat;
}

typedef struct {
    float* W1;
    float* W2;
    float* b1;
    float* b2;
} NeuralNetworkDevice;
NeuralNetworkDevice host_net;

__global__ void init_W1(float* W1) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = INPUT_SIZE * HIDDEN_SIZE;
    if (idx >= total) return;

    int inputIdx = idx % INPUT_SIZE;
    int hiddenIdx = idx / INPUT_SIZE;

    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    float rand_val = hiprand_uniform(&state);
    W1[hiddenIdx * INPUT_SIZE + inputIdx] = (rand_val - 0.5f) * 0.1f;
}

__global__ void init_W2(float* W2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = HIDDEN_SIZE * OUTPUT_SIZE;
    if (idx >= total) return;

    int hiddenIdx = idx % HIDDEN_SIZE;
    int outputIdx = idx / HIDDEN_SIZE;

    hiprandState state;
    hiprand_init(5678, idx, 0, &state);

    float rand_val = hiprand_uniform(&state);
    W2[outputIdx * HIDDEN_SIZE + hiddenIdx] = (rand_val - 0.5f) * 0.1f;
}

NeuralNetworkDevice* createNetworkOnDevice() {
    NeuralNetworkDevice* dev_net;

    hipMalloc((void**)&host_net.W1, sizeof(float) * HIDDEN_SIZE * INPUT_SIZE);
    hipMalloc((void**)&host_net.W2, sizeof(float) * OUTPUT_SIZE * HIDDEN_SIZE);
    hipMalloc((void**)&host_net.b1, sizeof(float) * HIDDEN_SIZE);
    hipMalloc((void**)&host_net.b2, sizeof(float) * OUTPUT_SIZE);
    hipMemset(host_net.b1, 0, sizeof(float) * HIDDEN_SIZE);
    hipMemset(host_net.b2, 0, sizeof(float) * OUTPUT_SIZE);

    int total_W1 = INPUT_SIZE * HIDDEN_SIZE;
    int total_W2 = HIDDEN_SIZE * OUTPUT_SIZE;

    int threads = 256;
    int blocks_W1 = (total_W1 + threads - 1) / threads;
    int blocks_W2 = (total_W2 + threads - 1) / threads;

    init_W1<<<blocks_W1, threads>>>(host_net.W1);
    init_W2<<<blocks_W2, threads>>>(host_net.W2);

    hipMalloc((void**)&dev_net, sizeof(NeuralNetworkDevice));
    hipMemcpy(dev_net, &host_net, sizeof(NeuralNetworkDevice), hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    return dev_net;
}


__global__ void init_hidden(NeuralNetworkDevice* net, float* hidden) {
    int idx = threadIdx.x;
    if (idx < HIDDEN_SIZE) {
        hidden[idx] = net->b1[idx];
    }
}

__global__ void hiddenLayerForwardBatch(NeuralNetworkDevice* net, float* input_batch, float* hidden_batch) {
    extern __shared__ float input_shared[];

    int batch_idx = blockIdx.x;
    int neuron_idx = threadIdx.x;

    if (neuron_idx < INPUT_SIZE) {
        input_shared[neuron_idx] = input_batch[batch_idx * INPUT_SIZE + neuron_idx];
    }
    __syncthreads();

    if (neuron_idx < HIDDEN_SIZE) {
        float sum = net->b1[neuron_idx];
        for (int j = 0; j < INPUT_SIZE; j++) {
            sum += input_shared[j] * net->W1[neuron_idx * INPUT_SIZE + j];
        }
        hidden_batch[batch_idx * HIDDEN_SIZE + neuron_idx] = sum;
    }
}

__global__ void relu_d(float* hidden) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < BATCH_SIZE * HIDDEN_SIZE && hidden[idx] < 0.0f)
        hidden[idx] = 0.0f;
}

__global__ void outputLayerForwardBatch(NeuralNetworkDevice* net, float* hidden_batch, float* output_batch) {
    extern __shared__ float hidden_shared[];

    int batch_idx = blockIdx.x;
    int neuron_idx = threadIdx.x;

    if (neuron_idx < HIDDEN_SIZE) {
        hidden_shared[neuron_idx] = hidden_batch[batch_idx * HIDDEN_SIZE + neuron_idx];
    }
    __syncthreads();

    if (neuron_idx < OUTPUT_SIZE) {
        float sum = net->b2[neuron_idx];
        for (int j = 0; j < HIDDEN_SIZE; j++) {
            sum += hidden_shared[j] * net->W2[neuron_idx * HIDDEN_SIZE + j];
        }
        output_batch[batch_idx * OUTPUT_SIZE + neuron_idx] = sum;
    }
}

__global__ void softmax_batch(float* output_batch) {
    int batch_idx = blockIdx.x;

    float max_val = -1e20f;
    for (int i = 0; i < OUTPUT_SIZE; ++i) {
        float val = output_batch[batch_idx * OUTPUT_SIZE + i];
        if (val > max_val) max_val = val;
    }

    float sum = 0.0f;
    for (int i = 0; i < OUTPUT_SIZE; ++i) {
        float val = expf(output_batch[batch_idx * OUTPUT_SIZE + i] - max_val);
        output_batch[batch_idx * OUTPUT_SIZE + i] = val;
        sum += val;
    }

    for (int i = 0; i < OUTPUT_SIZE; ++i) {
        output_batch[batch_idx * OUTPUT_SIZE + i] /= sum;
    }
}

void forward(NeuralNetworkDevice* net, float* hidden_batch, float* output_batch, float* input_batch, hipStream_t stream) {
    size_t shared_input = INPUT_SIZE * sizeof(float);
    hiddenLayerForwardBatch<<<BATCH_SIZE, max(HIDDEN_SIZE, INPUT_SIZE), shared_input, stream>>>(net, input_batch, hidden_batch);

    int reluThreads = 256;
    int reluBlocks = (BATCH_SIZE * HIDDEN_SIZE + reluThreads - 1) / reluThreads;
    relu_d<<<reluBlocks, reluThreads, 0, stream>>>(hidden_batch);

    size_t shared_hidden = HIDDEN_SIZE * sizeof(float);
    outputLayerForwardBatch<<<BATCH_SIZE, max(HIDDEN_SIZE, OUTPUT_SIZE), shared_hidden, stream>>>(net, hidden_batch, output_batch);

    softmax_batch<<<BATCH_SIZE, 1, 0, stream>>>(output_batch);
}

__global__
void layerGradientBatch(float* d_output_batch, float* output_batch, float* target_batch) {
    int batch_idx = blockIdx.x;
    int i = threadIdx.x;

    int offset = batch_idx * OUTPUT_SIZE + i;
    if (i < OUTPUT_SIZE)
        d_output_batch[offset] = output_batch[offset] - target_batch[offset];
}


__global__
void hiddenLayerGradientBatch(NeuralNetworkDevice* net, float* d_hidden_batch, float* hidden_batch, float* d_output_batch) {
    int batch_idx = blockIdx.x;
    int hidden_idx = threadIdx.x;

    float sum = 0.0f;
    for (int output_idx = 0; output_idx < OUTPUT_SIZE; ++output_idx) {
        int output_offset = batch_idx * OUTPUT_SIZE + output_idx;
        int w_idx = output_idx * HIDDEN_SIZE + hidden_idx;
        sum += net->W2[w_idx] * d_output_batch[output_offset];
    }

    int h_off = batch_idx * HIDDEN_SIZE + hidden_idx;
    float relu_grad = (hidden_batch[h_off] > 0.0f) ? 1.0f : 0.0f;
    d_hidden_batch[h_off] = sum * relu_grad;
}

__global__
void updateWeights1Batch(NeuralNetworkDevice* net, float* d_hidden_batch, float* input_batch, int batch_size) {
    int hidden_idx = blockIdx.x;
    int input_idx = threadIdx.x;

    float grad = 0.0f;
    float bias_grad = 0.0f;

    for (int b = 0; b < batch_size; b++) {
        int h_off = b * HIDDEN_SIZE + hidden_idx;
        int i_off = b * INPUT_SIZE + input_idx;
        grad += d_hidden_batch[h_off] * input_batch[i_off];
        if (input_idx == 0) bias_grad += d_hidden_batch[h_off];
    }

    int w_idx = hidden_idx * INPUT_SIZE + input_idx;
    atomicAdd(&net->W1[w_idx], -LEARNING_RATE * grad / batch_size);

    if (input_idx == 0) {
        atomicAdd(&net->b1[hidden_idx], -LEARNING_RATE * bias_grad / batch_size);
    }
}

__global__
void updateWeights2Batch(NeuralNetworkDevice* net, float* hidden_batch, float* d_output_batch, int batch_size) {
    int output_idx = blockIdx.x;
    int hidden_idx = threadIdx.x;

    float grad = 0.0f;
    float bias_grad = 0.0f;

    for (int b = 0; b < batch_size; b++) {
        int h_off = b * HIDDEN_SIZE + hidden_idx;
        int o_off = b * OUTPUT_SIZE + output_idx;
        grad += d_output_batch[o_off] * hidden_batch[h_off];
        if (hidden_idx == 0) bias_grad += d_output_batch[o_off];
    }

    int w_idx = output_idx * HIDDEN_SIZE + hidden_idx;
    atomicAdd(&net->W2[w_idx], -LEARNING_RATE * grad / batch_size);

    if (hidden_idx == 0) {
        atomicAdd(&net->b2[output_idx], -LEARNING_RATE * bias_grad / batch_size);
    }
}

void backward_batch(NeuralNetworkDevice* net, float* hidden_batch, float* output_batch, float* label_batch,
                    float* input_batch, int batch_size, hipStream_t stream) {
    float* d_output_batch;
    float* d_hidden_batch;

    hipMallocAsync(&d_output_batch, batch_size * OUTPUT_SIZE * sizeof(float), stream);
    hipMallocAsync(&d_hidden_batch, batch_size * HIDDEN_SIZE * sizeof(float), stream);

    hipMemsetAsync(d_output_batch, 0, batch_size * OUTPUT_SIZE * sizeof(float), stream);
    hipMemsetAsync(d_hidden_batch, 0, batch_size * HIDDEN_SIZE * sizeof(float), stream);

    dim3 grid1(batch_size);
    dim3 block1(OUTPUT_SIZE);
    layerGradientBatch<<<grid1, block1, 0, stream>>>(d_output_batch, output_batch, label_batch);

    dim3 grid2(batch_size);
    dim3 block2(HIDDEN_SIZE);
    hiddenLayerGradientBatch<<<grid2, block2, 0, stream>>>(net, d_hidden_batch, hidden_batch, d_output_batch);

    dim3 grid3(OUTPUT_SIZE);
    dim3 block3(HIDDEN_SIZE);
    updateWeights2Batch<<<grid3, block3, 0, stream>>>(net, hidden_batch, d_output_batch, batch_size);

    dim3 grid4(HIDDEN_SIZE);
    dim3 block4(INPUT_SIZE);
    updateWeights1Batch<<<grid4, block4, 0, stream>>>(net, d_hidden_batch, input_batch, batch_size);

    hipFreeAsync(d_output_batch, stream);
    hipFreeAsync(d_hidden_batch, stream);
}

void assignMemory(float** hidden, float** output, float** label) {
    hipMalloc((void**)hidden, HIDDEN_SIZE * sizeof(float));
    hipMalloc((void**)output, OUTPUT_SIZE * sizeof(float));
    hipMalloc((void**)label, OUTPUT_SIZE * sizeof(float));
}


void train(NeuralNetworkDevice* net_d, float** images, float** labels, int numImages) {
    size_t hidden_bytes = BATCH_SIZE * HIDDEN_SIZE * sizeof(float);
    size_t input_bytes  = BATCH_SIZE * INPUT_SIZE * sizeof(float);
    size_t output_bytes = BATCH_SIZE * OUTPUT_SIZE * sizeof(float);

    hipStream_t stream;
    hipStreamCreate(&stream);

    float *input_batch_d, *hidden_d, *output_d, *label_batch_d;
    hipMallocAsync(&input_batch_d, input_bytes, stream);
    hipMallocAsync(&output_d, output_bytes, stream);
    hipMallocAsync(&label_batch_d, output_bytes, stream);
    hipMallocAsync(&hidden_d, hidden_bytes, stream);

    float* input_host;
    float* label_host;
    float* output_host;
    hipHostMalloc(&input_host, input_bytes);
    hipHostMalloc(&label_host, output_bytes);
    hipHostMalloc(&output_host, output_bytes);

    clock_t total_start = clock();

    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        clock_t epoch_start = clock();
        float loss = 0.0f;
        int correct = 0;

        for (int i = 0; i < numImages; i += BATCH_SIZE) {
            int current_batch_size = min(BATCH_SIZE, numImages - i);

            for (int b = 0; b < current_batch_size; b++) {
                memcpy(input_host + b * INPUT_SIZE, images[i + b], INPUT_SIZE * sizeof(float));
                memcpy(label_host + b * OUTPUT_SIZE, labels[i + b], OUTPUT_SIZE * sizeof(float));
            }

            hipMemcpyAsync(input_batch_d, input_host, current_batch_size * INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice, stream);
            hipMemcpyAsync(label_batch_d, label_host, current_batch_size * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice, stream);

            forward(net_d, hidden_d, output_d, input_batch_d, stream);
            backward_batch(net_d, hidden_d, output_d, label_batch_d, input_batch_d, current_batch_size, stream);

            hipStreamSynchronize(stream);

            hipMemcpy(output_host, output_d, current_batch_size * OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost);

            for (int b = 0; b < current_batch_size; b++) {
                int idx = i + b;
                float* output_row = output_host + b * OUTPUT_SIZE;
                float* label_row = labels[idx];

                for (int k = 0; k < OUTPUT_SIZE; k++)
                    loss -= label_row[k] * logf(output_row[k] + 1e-12f);

                int pred = 0, actual = 0;
                for (int j = 0; j < OUTPUT_SIZE; j++) {
                    if (output_row[j] > output_row[pred]) pred = j;
                    if (label_row[j] > label_row[actual]) actual = j;
                }
                if (pred == actual) correct++;
            }
        }

        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
               epoch + 1, loss / numImages, (correct / (float)numImages) * 100, get_time(epoch_start));
    }

    printf("Total training time: %.3fs\n", get_time(total_start));

    hipStreamDestroy(stream);
    hipFreeAsync(input_batch_d, stream);
    hipFreeAsync(hidden_d, stream);
    hipFreeAsync(output_d, stream);
    hipFreeAsync(label_batch_d, stream);

    hipHostFree(input_host);
    hipHostFree(label_host);
    hipHostFree(output_host);
}

void evaluate(NeuralNetworkDevice* net_d, float** images, float** labels, int numImages) {
    size_t input_bytes = INPUT_SIZE * sizeof(float);
    size_t hidden_bytes = HIDDEN_SIZE * sizeof(float);
    size_t output_bytes = OUTPUT_SIZE * sizeof(float);

    hipStream_t eval_stream;
    hipStreamCreate(&eval_stream);

    float *input_batch_d, *hidden_d, *output_d, *label_d;
    hipMallocAsync(&input_batch_d, input_bytes, eval_stream);
    hipMallocAsync(&hidden_d, hidden_bytes, eval_stream);
    hipMallocAsync(&output_d, output_bytes, eval_stream);
    hipMallocAsync(&label_d, output_bytes, eval_stream);

    float* output_host;
    hipHostMalloc(&output_host, output_bytes);

    int correct = 0;

    for (int i = 0; i < numImages; i++) {
        hipMemcpyAsync(input_batch_d, images[i], input_bytes, hipMemcpyHostToDevice, eval_stream);
        forward(net_d, hidden_d, output_d, input_batch_d, eval_stream);
        hipStreamSynchronize(eval_stream);
        hipMemcpy(output_host, output_d, output_bytes, hipMemcpyDeviceToHost);

        int pred = 0, actual = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            if (output_host[j] > output_host[pred]) pred = j;
            if (labels[i][j] > labels[i][actual]) actual = j;
        }
        if (pred == actual) correct++;
    }

    printf("Test Accuracy: %.2f%%\n", (correct / (float)numImages) * 100.0f);

    hipFreeAsync(input_batch_d, eval_stream);
    hipFreeAsync(hidden_d, eval_stream);
    hipFreeAsync(output_d, eval_stream);
    hipFreeAsync(label_d, eval_stream);
    hipHostFree(output_host);
    hipStreamDestroy(eval_stream);
}


float** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 16, SEEK_SET);
    float** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;
            if (fread(&pixel, sizeof(unsigned char), 1, file) != 1) {
                fprintf(stderr, "Error: Failed to read pixel\n");
                fclose(file);
                exit(EXIT_FAILURE);
            }
            images[i][j] = pixel / 255.0f;
        }
    }
    fclose(file);
    return images;
}

float** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) {
        printf("Error opening %s\n", filename);
        exit(1);
    }
    fseek(file, 8, SEEK_SET);
    float** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        if (fread(&label, sizeof(unsigned char), 1, file) != 1) {
            fprintf(stderr, "Error: Failed to read label\n");
            fclose(file);
            exit(EXIT_FAILURE);
        }
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            labels[i][j] = (j == label) ? 1.0f : 0.0f;
        }
    }
    fclose(file);
    return labels;
}

int main() {
    printf("MNIST Neural Network with Tensor Cores\n\n");

    float** train_images = loadMNISTImages("../../data/train-images.idx3-ubyte", 60000);
    float** train_labels = loadMNISTLabels("../../data/train-labels.idx1-ubyte", 60000);
    float** test_images = loadMNISTImages("../../data/t10k-images.idx3-ubyte", 10000);
    float** test_labels = loadMNISTLabels("../../data/t10k-labels.idx1-ubyte", 10000);
    NeuralNetworkDevice* net_d = createNetworkOnDevice();

    train(net_d, train_images, train_labels, 60000);
    evaluate(net_d, test_images, test_labels, 10000);

    return 0;
}
